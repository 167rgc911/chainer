#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_rnn.h"

#include <algorithm>
#include <cstdint>
#include <functional>
#include <utility>
#include <vector>
#include <iostream>

#include <nonstd/optional.hpp>

#include "chainerx/array.h"
#include "chainerx/cuda/copy_data.cuh"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/cuda_backend.h"
#include "chainerx/cuda/cuda_device.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/hipDNN.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/hash_combine.h"
#include "chainerx/kernels/connection.h"
#include "chainerx/macro.h"
#include "chainerx/routines/connection.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/manipulation.h"
#include "chainerx/shape.h"
#include "chainerx/stack_vector.h"

namespace chainerx {
namespace cuda {
namespace cuda_internal {


float *reserve;

size_t reserve_size;


std::vector<Array> split(const Array& ary, std::vector<int64_t> indices, int8_t axis) {
    const Shape& in_shape = ary.shape();
    int8_t axis_norm = internal::NormalizeAxis(axis, ary.ndim());
    int64_t in_dim = in_shape[axis_norm];

    // Wrap negative indices.
    std::transform(
            indices.begin(), indices.end(), indices.begin(), [in_dim](int64_t index) { return index >= 0 ? index : index + in_dim; });
    indices.emplace_back(in_dim);

    Shape out_shape = in_shape;
    int64_t out_stride = ary.strides()[axis_norm];
    int64_t out_offset = ary.offset();
    int64_t slice_start = 0;
    bool is_empty = ary.GetTotalSize() == 0;

    std::vector<Array> out{};
    out.reserve(indices.size());

    for (int64_t index : indices) {
        int64_t slice_stop = std::min(in_dim, std::max(int64_t{0}, index));
        int64_t slice_step = slice_stop - slice_start;

        // Update the dimension of interest in the output shape.
        out_shape[axis_norm] = std::max(int64_t{0}, slice_step);

        out.emplace_back(internal::MakeArray(out_shape, ary.strides(), ary.dtype(), ary.device(), ary.data(), out_offset));

        // Empty arrays should all have offsets of 0 to e.g. avoid out-of-memory errors.
        if (!is_empty) {
            out_offset += out_stride * slice_step;
        }

        slice_start = slice_stop;
    }
    return out;
} 
void weights_forward(
    CudnnHandle& handle,
    hipdnnRNNDescriptor_t rnn_desc,
    std::vector<std::vector<Array>> ws,
    std::vector<std::vector<Array>> bs,
    int n_layers,
    int num_directions,
    hipdnnTensorDescriptor_t x_desc,
    hipdnnFilterDescriptor_t w_desc,
    Array& w
    ) {
    for(int layer = 0 ; layer < n_layers; layer++) {
        for(int8_t di = 0; di < num_directions; di++) {
            for(uint lin_layer_id = 0; lin_layer_id < ws[0].size(); lin_layer_id++) {
                int64_t index = num_directions * layer + di;
                
                hipdnnFilterDescriptor_t linlayermatdesc;
                hipdnnCreateFilterDescriptor(&linlayermatdesc);
                float* m_offset;
                handle.Call(
                    hipdnnGetRNNLinLayerMatrixParams,
                    rnn_desc,
                    layer,
                    x_desc,
                    w_desc,
                    internal::GetRawOffsetData(AsContiguous(w)),
                    lin_layer_id,
                    linlayermatdesc,
                    (void**)&m_offset         
                );
                hipdnnDataType_t dataType;
                hipdnnTensorFormat_t format;
                int nbDims;
                int filterDimA[3];


                hipdnnGetFilterNdDescriptor(
                linlayermatdesc,
                3,
                &dataType,
                &format,
                &nbDims,
                filterDimA
                );
                ws[index][lin_layer_id] = ws[index][lin_layer_id].AsType(Dtype::kFloat32);
                initGPUData(m_offset, filterDimA[0] * filterDimA[1] * filterDimA[2], (float*)internal::GetRawOffsetData(AsContiguous(ws[index][lin_layer_id])));
                hipdnnDestroyFilterDescriptor(linlayermatdesc);
                

                hipdnnFilterDescriptor_t linlayerbiasdesc;
                hipdnnCreateFilterDescriptor(&linlayerbiasdesc);
                float* b_offset;
                handle.Call(
                    hipdnnGetRNNLinLayerBiasParams,
                    rnn_desc,
                    layer,
                    x_desc,
                    w_desc,
                    internal::GetRawOffsetData(AsContiguous(w)),
                    lin_layer_id,
                    linlayerbiasdesc,
                    (void**)&b_offset
                );
                hipdnnGetFilterNdDescriptor(
                                        linlayerbiasdesc,
                                        3,
                                        &dataType,
                                        &format,
                                        &nbDims,
                                        filterDimA
                                        );
                bs[index][lin_layer_id] = bs[index][lin_layer_id].AsType(Dtype::kFloat32);
                initGPUData(b_offset, filterDimA[0] * filterDimA[1] * filterDimA[2], (float*)internal::GetRawOffsetData(AsContiguous(bs[index][lin_layer_id])));
                hipdnnDestroyFilterDescriptor(linlayerbiasdesc);
            }
        }
    }
}

std::vector<std::vector<Array>> CudaRnn::n_step_rnn(
        CudaDevice& device,
        int64_t n_layers,
        Array hx,
        Array cx,
        const std::vector<std::vector<Array>>& ws,
        const std::vector<std::vector<Array>>& bs,
        std::vector<Array>& xs,
        const int8_t bidirectional,
        const int8_t mode) {

    CudaSetDeviceScope scope{device.index()};
    auto& backend = static_cast<CudaBackend&>(device.backend());  // NOLINT
    Dtype type = hx.dtype();
    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(device);

    CudnnHandle& handle = device_internals.cudnn_handle();

    const auto input_dim = xs[0].shape()[1];
    const auto hidden_dim = hx.shape()[2];
    const auto num_directions = bidirectional == 1 ? 2 : 1;
    const auto num_layers = n_layers;
    const auto rnn_direction = bidirectional == 1 ? HIPDNN_BIDIRECTIONAL : HIPDNN_UNIDIRECTIONAL;
    const auto rnn_mode = mode == 1? HIPDNN_LSTM : HIPDNN_GRU ;
    const auto rnn_input = HIPDNN_LINEAR_INPUT;
    hipdnnDropoutDescriptor_t dropoutDesc;
    hipdnnRNNDescriptor_t rnn_desc;

    unsigned long long seed = 1337ull; 

   
   hipdnnCreateDropoutDescriptor(&dropoutDesc);

   
   size_t stateSize;
   void *states;
   handle.Call(hipdnnDropoutGetStatesSize, &stateSize);

   hipMalloc(&states, stateSize);

   hipdnnSetDropoutDescriptor(dropoutDesc,
                             handle.handle(),
                             0,
                             states,
                             stateSize,
                             seed);

    hipdnnCreateRNNDescriptor(&rnn_desc);
    handle.Call(
        hipdnnSetRNNDescriptor,
        rnn_desc,
        hidden_dim,
        num_layers,
        dropoutDesc,
        rnn_input,
        rnn_direction,
        rnn_mode,
        HIPDNN_RNN_ALGO_STANDARD,
        HIPDNN_DATA_FLOAT
    );
    
    hipdnnTensorDescriptor_t *x_desc, *y_desc;
    x_desc = (hipdnnTensorDescriptor_t*)malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t));
    y_desc = (hipdnnTensorDescriptor_t*)malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t));
    std::vector<CudnnTensorDescriptor> xs_desc;
    std::vector<CudnnTensorDescriptor> ys_desc;
    std::vector<Array> ys;
    for(uint i = 0; i < xs.size(); i++) {
        Shape xs_shape{xs[i].shape()[0], xs[i].shape()[1], 1};
        Shape ys_shape{xs[i].shape()[0], num_directions * hidden_dim, 1};
        xs[i] = xs[i].AsType(Dtype::kFloat32);
        ys.push_back(Empty({xs[i].shape()[0], num_directions * hidden_dim}, xs[i].dtype(), xs[i].device()));
        xs_desc.push_back(CudnnTensorDescriptor(AsContiguous(xs[i]).Reshape(xs_shape)));
        ys_desc.push_back(CudnnTensorDescriptor(AsContiguous(ys[i]).Reshape( ys_shape)));
        x_desc[i] = *xs_desc[i];
        y_desc[i] = *ys_desc[i];
    }

    
    Array x = Concatenate(xs, 0);
    Array y = Concatenate(ys, 0);


    size_t weight_size;
    handle.Call(
        hipdnnGetRNNParamsSize,
        rnn_desc,
        x_desc[0],
        &weight_size,
        HIPDNN_DATA_FLOAT
    );



    Array w = Empty({(int)weight_size / 4, 1, 1}, x.dtype(), x.device());
    CudnnFilterDescriptor wDesc{w};

    weights_forward(handle, rnn_desc, ws, bs, n_layers, num_directions, x_desc[0], *wDesc, w);

    
    void *workspace;
    size_t workSize;
    handle.Call(hipdnnGetRNNWorkspaceSize, rnn_desc, xs.size(), x_desc, &workSize);
    hipMalloc((void**)&workspace, workSize);

    handle.Call(hipdnnGetRNNTrainingReserveSize, rnn_desc, xs.size(), x_desc, &reserve_size);
    hipMallocManaged((void**)&reserve, reserve_size);
    hx = hx.AsType(Dtype::kFloat32);
    cx = cx.AsType(Dtype::kFloat32);
    Array hy = Empty(hx.shape(), hx.dtype(), hx.device());
    Array cy = Empty(cx.shape(), cx.dtype(), cx.device());
    CudnnTensorDescriptor hxDesc{AsContiguous(hx)};
    CudnnTensorDescriptor cxDesc{AsContiguous(cx)};

    CudnnTensorDescriptor hyDesc{hy};
    CudnnTensorDescriptor cyDesc{cy};

    handle.Call(
        hipdnnRNNForwardTraining,
        rnn_desc,
        xs.size(),
        x_desc,
        internal::GetRawOffsetData(AsContiguous(x)),
        *hxDesc,
        internal::GetRawOffsetData(AsContiguous(hx)),
        *cxDesc,
        internal::GetRawOffsetData(AsContiguous(cx)),
        *wDesc,
        internal::GetRawOffsetData(w),
        y_desc,
        internal::GetRawOffsetData(y),
        *hyDesc,
        internal::GetRawOffsetData(hy),
        *cyDesc,
        internal::GetRawOffsetData(cy),
        workspace,
        workSize,
        reserve,
        reserve_size
    );

    
    std::vector<int64_t> split_indices;
    for(uint i = 0; i < xs.size() - 1; i++){
        if ( i != 0 ) {
            split_indices.push_back(split_indices[i-1] + xs[i].shape()[0]);
        } else {
            split_indices.push_back(xs[i].shape()[0]);
        }
    }
    y = y.AsType(type);
    ys = split(y, split_indices, 0);

    std::vector<Array> out_states;
    out_states.push_back(hy.AsType(type));
    out_states.push_back(cy.AsType(type));
    std::vector<std::vector<Array>> ret;
    ret.push_back(out_states);
    ret.push_back(ys);
    return ret;
}

std::vector<Array> weights_backward(
    CudaDevice &device,
    hipdnnRNNDescriptor_t& rnn_desc,
    hipdnnTensorDescriptor_t dummy_x_desc,
    hipdnnFilterDescriptor_t w_desc,
    Array w,
    const std::vector<std::vector<Array>> ws,
    const std::vector<std::vector<Array>> bs,
    int64_t n_layers,
    int64_t num_directions,
    Dtype type
    ) {
    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(device);
    std::vector<Array> ret;
    CudnnHandle& handle = device_internals.cudnn_handle();
    
    for(int64_t layer = 0 ; layer < n_layers; layer++) {
        for(int8_t di = 0; di < num_directions; di++) {
            for(uint lin_layer_id = 0; lin_layer_id < ws[0].size(); lin_layer_id++) {
                int64_t index = num_directions * layer + di;
                
                
                hipdnnFilterDescriptor_t linlayermatdesc;
                hipdnnCreateFilterDescriptor(&linlayermatdesc);
                float* m_offset;
                handle.Call(
                    hipdnnGetRNNLinLayerMatrixParams,
                    rnn_desc,
                    layer,
                    dummy_x_desc,
                    w_desc,
                    internal::GetRawOffsetData(w),
                    lin_layer_id,
                    linlayermatdesc,
                    (void**)&m_offset
                );
                
                Array m = Empty(ws[index][lin_layer_id].shape(), ws[index][lin_layer_id].dtype(), ws[index][lin_layer_id].device());
                hipdnnDataType_t dataType;
                hipdnnTensorFormat_t format;
                int nbDims;
                int filterDimA[3];


                hipdnnGetFilterNdDescriptor(
                linlayermatdesc,
                3,
                &dataType,
                &format,
                &nbDims,
                filterDimA
                );
                initGPUData((float*)internal::GetRawOffsetData(m), filterDimA[0] * filterDimA[1] * filterDimA[2], m_offset);
                hipdnnDestroyFilterDescriptor(linlayermatdesc);
                ret.push_back(m.AsType(type));
                
                hipdnnFilterDescriptor_t linlayerbiasdesc;
                hipdnnCreateFilterDescriptor(&linlayerbiasdesc);
                float* b_offset;
                handle.Call(
                    hipdnnGetRNNLinLayerBiasParams,
                    rnn_desc,
                    layer,
                    dummy_x_desc,
                    w_desc,
                    internal::GetRawOffsetData(w),
                    lin_layer_id,
                    linlayerbiasdesc,
                    (void**)&b_offset
                );
                
                Array b = Empty(bs[index][lin_layer_id].shape(), bs[index][lin_layer_id].dtype(), bs[index][lin_layer_id].device());
                hipdnnGetFilterNdDescriptor(
                                        linlayerbiasdesc,
                                        3,
                                        &dataType,
                                        &format,
                                        &nbDims,
                                        filterDimA
                                        );
                initGPUData((float*)internal::GetRawOffsetData(b), filterDimA[0] * filterDimA[1] * filterDimA[2], b_offset);
                hipdnnDestroyFilterDescriptor(linlayerbiasdesc);
                
                ret.push_back(b.AsType(type));
            }
        }
    }
    return ret;
}

std::vector<std::vector<Array>> CudaRnn::n_step_rnn_backward(
        CudaDevice& device,
        int64_t n_layers,
        Array hx,
        Array cx,
        const std::vector<std::vector<Array>>& ws,
        const std::vector<std::vector<Array>>& bs,
        std::vector<Array>& xs,
        Array dhy,
        Array dcy,
        std::vector<Array> ys,
        std::vector<Array> dys,
        const int8_t bidirectional,
        const int8_t mode) {
    CudaSetDeviceScope scope{device.index()};

    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(device);

    CudnnHandle& handle = device_internals.cudnn_handle();
    Dtype type = hx.dtype();
    const auto input_dim = xs[0].shape()[1];
    const auto hidden_dim = hx.shape()[2];
    const auto num_directions = bidirectional == 1 ? 2 : 1;
    const auto num_layers = n_layers;
    const auto rnn_direction = bidirectional == 1 ? HIPDNN_BIDIRECTIONAL : HIPDNN_UNIDIRECTIONAL;
    const auto rnn_mode = mode == 1 ? HIPDNN_LSTM : HIPDNN_GRU;
    const auto rnn_input = HIPDNN_LINEAR_INPUT;
    unsigned long long seed = 1337ull; 

   hipdnnDropoutDescriptor_t dropoutDesc;
   hipdnnCreateDropoutDescriptor(&dropoutDesc);

   
   size_t stateSize;
   void *states;
   handle.Call(hipdnnDropoutGetStatesSize, &stateSize);

   hipMalloc(&states, stateSize);

   hipdnnSetDropoutDescriptor(dropoutDesc,
                             handle.handle(),
                             0,
                             states,
                             stateSize,
                             seed);


    hipdnnRNNDescriptor_t rnn_desc;
    hipdnnCreateRNNDescriptor(&rnn_desc);
    handle.Call(
        hipdnnSetRNNDescriptor,
        rnn_desc,
        hidden_dim,
        num_layers,
        dropoutDesc,
        rnn_input,
        rnn_direction,
        rnn_mode,
        HIPDNN_RNN_ALGO_STANDARD,
        HIPDNN_DATA_FLOAT
    );
    std::vector<Array> dxs;
    std::vector<CudnnTensorDescriptor> xsDesc, dxsDesc, ysDesc, dysDesc; 
    hipdnnTensorDescriptor_t *xs_desc = (hipdnnTensorDescriptor_t*)malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t));
    hipdnnTensorDescriptor_t *dxs_desc = (hipdnnTensorDescriptor_t*)malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t));
    hipdnnTensorDescriptor_t *ys_desc = (hipdnnTensorDescriptor_t*)malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t));
    hipdnnTensorDescriptor_t *dys_desc = (hipdnnTensorDescriptor_t*)malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t)); 

    for(uint i = 0; i < xs.size(); i++) {
        Shape xs_shape{xs[i].shape()[0], xs[i].shape()[1], 1};
        Shape ys_shape{ys[i].shape()[0], ys[i].shape()[1], 1};
        xs[i] = xs[i].AsType(Dtype::kFloat32);
        ys[i] = ys[i].AsType(Dtype::kFloat32);
        dys[i] = dys[i].AsType(Dtype::kFloat32);
        xsDesc.push_back(CudnnTensorDescriptor(AsContiguous(xs[i]).Reshape(xs_shape)));
        xs_desc[i] = *xsDesc[i];
        dxs.push_back(Empty(xs[i].shape(), xs[i].dtype(), xs[i].device()));
        dxsDesc.push_back(CudnnTensorDescriptor(AsContiguous(dxs[i]).Reshape(xs_shape)));
        dxs_desc[i] = *dxsDesc[i];
        ysDesc.push_back(CudnnTensorDescriptor(AsContiguous(ys[i]).Reshape(ys_shape)));
        ys_desc[i] = *ysDesc[i];
        dysDesc.push_back(CudnnTensorDescriptor(AsContiguous(dys[i]).Reshape(ys_shape)));
        dys_desc[i] = *dysDesc[i];
    }
    Array dx = Concatenate(dxs, 0);
    Array x = Concatenate(xs, 0);
    Array y = Concatenate(ys, 0);
    Array dy = Concatenate(dys, 0);
    size_t weight_size;
    handle.Call(
        hipdnnGetRNNParamsSize,
        rnn_desc,
        xs_desc[0],
        &weight_size,
        HIPDNN_DATA_FLOAT
    );
    Array w = Empty({(int)weight_size / 4, 1, 1}, x.dtype(), x.device());
    CudnnFilterDescriptor wDesc{w};
    
    
    weights_forward(handle, rnn_desc, ws, bs, n_layers, num_directions, xs_desc[0], *wDesc, w);

    void *workspace;
    size_t workSize;
    handle.Call(hipdnnGetRNNWorkspaceSize, rnn_desc, xs.size(), xs_desc, &workSize);
    hipMalloc((void**)&workspace, workSize);

    hx = hx.AsType(Dtype::kFloat32);
    cx = cx.AsType(Dtype::kFloat32);
    dhy = dhy.AsType(Dtype::kFloat32);
    dcy = dcy.AsType(Dtype::kFloat32);

    CudnnTensorDescriptor hx_desc{AsContiguous(hx)};
    CudnnTensorDescriptor cx_desc{AsContiguous(cx)};
    CudnnTensorDescriptor dhy_desc{AsContiguous(dhy)};
    CudnnTensorDescriptor dcy_desc{AsContiguous(dcy)};

    Array dhx = Empty(hx.shape(), hx.dtype(), hx.device());
    CudnnTensorDescriptor dhx_desc{dhx};
    Array dcx = Empty(cx.shape(), cx.dtype(), cx.device());
    CudnnTensorDescriptor dcx_desc{dcx};
    
    handle.Call(
        hipdnnRNNBackwardData,
        rnn_desc,
        xs.size(),
        ys_desc,
        internal::GetRawOffsetData(AsContiguous(y)),
        dys_desc,
        internal::GetRawOffsetData(AsContiguous(dy)),
        *dhy_desc,
        internal::GetRawOffsetData(AsContiguous(dhy)),
        *dcy_desc,
        internal::GetRawOffsetData(AsContiguous(dcy)),
        *wDesc,
        internal::GetRawOffsetData(w),
        *hx_desc,
        internal::GetRawOffsetData(AsContiguous(hx)),
        *cx_desc,
        internal::GetRawOffsetData(AsContiguous(cx)),
        dxs_desc,
        internal::GetRawOffsetData(AsContiguous(dx)),
        *dhx_desc,
        internal::GetRawOffsetData(dcx),
        *dcx_desc,
        internal::GetRawOffsetData(dhx),
        workspace,
        workSize,
        reserve,
        reserve_size
    );

    Array dw = Empty({(int)weight_size / 4, 1, 1}, hx.dtype(), hx.device());
    CudnnFilterDescriptor dwDesc{AsContiguous(dw)};
    
    handle.Call(
        hipdnnRNNBackwardWeights,
        rnn_desc,
        xs.size(),
        xs_desc,
        internal::GetRawOffsetData(AsContiguous(x)),
        *hx_desc,
        internal::GetRawOffsetData(AsContiguous(hx)),
        ys_desc,
        internal::GetRawOffsetData(AsContiguous(y)),
        workspace,
        workSize,
        *dwDesc,
        internal::GetRawOffsetData(dw),
        reserve,
        reserve_size
    );

    std::vector<int64_t> split_indices;
    for(uint i = 0; i < xs.size() - 1; i++){
        if ( i != 0 ) {
            split_indices.push_back(split_indices[i - 1] + xs[i].shape()[0]);
        } else {
            split_indices.push_back(xs[i].shape()[0]);
        }
    }
    dx = dx.AsType(type);
    dxs = split(dx, split_indices, 0);
    std::vector<Array> state;
    state.push_back(dhx.AsType(type));
    state.push_back(dcx.AsType(type));
    std::vector<std::vector<Array>> ret;
    ret.push_back(state);
    ret.push_back(weights_backward(device, rnn_desc, dxs_desc[0], *dwDesc, dw, ws, bs, n_layers, num_directions, type));
    ret.push_back(dxs);
    return ret;

}
}  // namespace cuda_internal
}  // namespace cuda
}  // namespace chainerx
