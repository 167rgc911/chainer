#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cstdint>
#include <mutex>
#include <type_traits>

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cuda_fp16.hpp>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/backend.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/hipblas.h"
#include "chainerx/cuda/cusolver.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/float16.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/linalg.h"
#include "chainerx/kernels/math.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/macro.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/linalg.h"
#include "chainerx/routines/math.h"

namespace chainerx {
namespace cuda {

class CudaQRKernel : public QRKernel {
public:
    std::tuple<Array, Array> Call(const Array& a, QRMode mode = QRMode::reduced) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);

        throw NotImplementedError("QR decomposition is not yet implemented for cuda device");

        if (mode != QRMode::reduced) {
            throw NotImplementedError{"Modes other than reduce are not implemented yet"};
        }

        int m = a.shape()[0];
        int n = a.shape()[1];
        int lda = std::min(m, n);

        Array Q = Empty(Shape({m, n}), dtype, device);
        Array R = Empty(Shape({n, n}), dtype, device);
        Array tau = Empty(Shape({n, 1}), dtype, device);

        auto qr_impl = [&](auto pt, auto geqrf_bufferSize, auto orgqr_bufferSize, auto geqrf, auto orgqr) {
            using T = typename decltype(pt)::type;
            cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

            T* a_ptr = static_cast<T*>(internal::GetRawOffsetData(a));
            T* q_ptr = static_cast<T*>(internal::GetRawOffsetData(Q));
            T* r_ptr = static_cast<T*>(internal::GetRawOffsetData(R));
            T* tau_ptr = static_cast<T*>(internal::GetRawOffsetData(tau));

            int *devInfo;
            CheckCudaError(hipMalloc(&devInfo, sizeof(int)));

            int lwork_geqrf = 0;
            int lwork_orgqr = 0;
            int lwork = 0;

            device_internals.cusolver_handle().Call(
                geqrf_bufferSize,
                m, n, a_ptr, lda, &lwork_geqrf);

            device_internals.cusolver_handle().Call(
                orgqr_bufferSize,
                m, n, n, a_ptr, lda, tau_ptr, &lwork_orgqr);

            lwork = (lwork_geqrf > lwork_orgqr) ? lwork_geqrf : lwork_orgqr;

            Array work = Empty(Shape({lwork, 1}), dtype, device);
            T* work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

            device_internals.cusolver_handle().Call(
                geqrf,
                m, n, a_ptr, lda, tau_ptr, work_ptr, lwork, devInfo);

            int devInfo_h = 0;
            CheckCudaError(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (devInfo_h != 0) {
                throw ChainerxError{"Unsuccessfull geqrf (QR) execution. Info = ", devInfo_h};
            }

            device_internals.cusolver_handle().Call(
                orgqr,
                m, n, n, a_ptr, lda, tau_ptr, work_ptr, lwork, devInfo);

            CheckCudaError(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (devInfo_h != 0) {
                throw ChainerxError{"Unsuccessfull orgqr (QR) execution. Info = ", devInfo_h};
            }
        };

        if (a.dtype() == Dtype::kFloat32) {
            qr_impl(PrimitiveType<float>{}, hipsolverDnSgeqrf_bufferSize, hipsolverDnSorgqr_bufferSize, hipsolverDnSgeqrf, hipsolverDnSorgqr);
        } else {
            CHAINERX_ASSERT(a.dtype() == Dtype::kFloat64);
            qr_impl(PrimitiveType<double>{}, hipsolverDnDgeqrf_bufferSize, hipsolverDnDorgqr_bufferSize, hipsolverDnDgeqrf, hipsolverDnDorgqr);
        }

        return std::make_tuple(std::move(Q), std::move(R));

    }
};

CHAINERX_CUDA_REGISTER_KERNEL(QRKernel, CudaQRKernel);

}  // namespace cuda
}  // namespace chainerx
