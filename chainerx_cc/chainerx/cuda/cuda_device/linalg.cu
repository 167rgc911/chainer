#include "chainerx/cuda/cuda_device.h"

#include <cstdint>
#include <mutex>
#include <type_traits>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <cuda_fp16.hpp>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/backend.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/hipblas.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/cusolver.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/float16.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/linalg.h"
#include "chainerx/kernels/math.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/macro.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/indexing.h"
#include "chainerx/routines/linalg.h"
#include "chainerx/routines/math.h"

namespace chainerx {
namespace cuda {

class CudaSyevdKernel : public SyevdKernel {
public:
    std::tuple<Array, Array> Call(const Array& a, const std::string& UPLO, bool compute_eigen_vector) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);

        Array v = Empty(a.shape(), dtype, device);
        device.backend().CallKernel<CopyKernel>(a.Transpose(), v);

        int64_t m = a.shape()[0];
        int64_t lda = a.shape()[1];

        Array w = Empty(Shape({m}), dtype, device);

        auto syevd_impl = [&](auto pt, auto syevd_bufferSize, auto syevd) -> std::tuple<Array, Array> {
            using T = typename decltype(pt)::type;
            cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

            T* v_ptr = static_cast<T*>(internal::GetRawOffsetData(v));
            T* w_ptr = static_cast<T*>(internal::GetRawOffsetData(w));

            hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
            if (compute_eigen_vector) {
                jobz = HIPSOLVER_EIG_MODE_VECTOR;
            }

            hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
            if (UPLO == "U") {
                uplo = HIPBLAS_FILL_MODE_UPPER;
            }

            int buffersize = 0;
            device_internals.cusolverdn_handle().Call(syevd_bufferSize, jobz, uplo, m, v_ptr, lda, w_ptr, &buffersize);

            Array work = Empty(Shape({buffersize}), dtype, device);
            T* work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

            int* devInfo;
            CheckCudaError(hipMalloc(&devInfo, sizeof(int)));

            device_internals.cusolverdn_handle().Call(syevd, jobz, uplo, m, v_ptr, lda, w_ptr, work_ptr, buffersize, devInfo);

            int devInfo_h = 0;
            CheckCudaError(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (devInfo_h != 0) {
                throw ChainerxError{"Unsuccessfull syevd (Eigen Decomposition) execution. Info = ", devInfo_h};
            }

            return std::make_tuple(std::move(w), std::move(v.Transpose().Copy()));
        };

        switch (a.dtype()) {
            case Dtype::kFloat16:
                throw DtypeError{"Half-precision (float16) is not supported by eigen decomposition"};
                break;
            case Dtype::kFloat32:
                return syevd_impl(PrimitiveType<float>{}, hipsolverDnSsyevd_bufferSize, hipsolverDnSsyevd);
                break;
            case Dtype::kFloat64:
                return syevd_impl(PrimitiveType<double>{}, hipsolverDnDsyevd_bufferSize, hipsolverDnDsyevd);
                break;
            default:
                CHAINERX_NEVER_REACH();
        }
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SyevdKernel, CudaSyevdKernel);

}  // namespace cuda
}  // namespace chainerx
