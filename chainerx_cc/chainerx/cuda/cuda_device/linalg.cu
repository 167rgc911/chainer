#include "chainerx/cuda/cuda_device.h"

#include <cstdint>
#include <mutex>
#include <type_traits>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <cuda_fp16.hpp>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/backend.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/hipblas.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/cusolver.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/float16.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/linalg.h"
#include "chainerx/kernels/math.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/macro.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/linalg.h"
#include "chainerx/routines/math.h"

namespace chainerx {
namespace cuda {

class CudaSolveKernel : public SolveKernel {
public:
    void Call(const Array& a, const Array& b, const Array& out) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        if (a.ndim() != 2) {
            throw DimensionError{"ChainerX solve supports only 2-dimensional arrays."};
        }
        if (a.shape()[0] != a.shape()[1]) {
            throw DimensionError{"Matrix is not square."};
        }

        auto solve_impl = [&](auto pt, auto getrf_bufferSize, auto getrf, auto getrs) {
            using T = typename decltype(pt)::type;
            cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

            Array lu_matrix = Empty(a.shape(), dtype, device);
            device.backend().CallKernel<CopyKernel>(a.Transpose(), lu_matrix);
            T* lu_ptr = static_cast<T*>(internal::GetRawOffsetData(lu_matrix));

            int m = a.shape()[0];

            Array ipiv = Empty(Shape({m}), Dtype::kInt32, device);
            int* ipiv_ptr = static_cast<int*>(internal::GetRawOffsetData(ipiv));

            int buffersize = 0;
            device_internals.cusolver_handle().Call(getrf_bufferSize, m, m, lu_ptr, m, &buffersize);

            Array work = Empty(Shape({buffersize}), dtype, device);
            T* work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

            int* devInfo;
            CheckCudaError(hipMalloc(&devInfo, sizeof(int)));

            device_internals.cusolver_handle().Call(getrf, m, m, lu_ptr, m, work_ptr, ipiv_ptr, devInfo);

            int devInfo_h = 0;
            CheckCudaError(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (devInfo_h != 0) {
                throw ChainerxError{"Unsuccessfull getrf (LU) execution. Info = ", devInfo_h};
            }

            device.backend().CallKernel<CopyKernel>(b, out);
            T* out_ptr = static_cast<T*>(internal::GetRawOffsetData(out));

            device_internals.cusolver_handle().Call(getrs, HIPBLAS_OP_N, m, m, lu_ptr, m, ipiv_ptr, out_ptr, m, devInfo);

            CheckCudaError(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (devInfo_h != 0) {
                throw ChainerxError{"Unsuccessfull getrs (Solve) execution. Info = ", devInfo_h};
            }
        };

        if (a.dtype() == Dtype::kFloat32) {
            solve_impl(PrimitiveType<float>{}, hipsolverDnSgetrf_bufferSize, hipsolverDnSgetrf, hipsolverDnSgetrs);
        } else {
            CHAINERX_ASSERT(a.dtype() == Dtype::kFloat64);
            solve_impl(PrimitiveType<double>{}, hipsolverDnDgetrf_bufferSize, hipsolverDnDgetrf, hipsolverDnDgetrs);
        }
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SolveKernel, CudaSolveKernel);

class CudaInverseKernel : public InverseKernel {
public:
    void Call(const Array& a, const Array& out) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        if (a.ndim() != 2) {
            throw DimensionError{"ChainerX inverse supports only 2-dimensional arrays."};
        }
        if (a.shape()[0] != a.shape()[1]) {
            throw DimensionError{"Matrix is not square."};
        }

        // There is LAPACK routine ``getri`` for computing the inverse of an LU-factored matrix,
        // but cuSOLVER does not have it implemented, therefore inverse is obtained with ``getrs``
        // inv(A) == solve(A, Identity)
        Array b = Identity(a.shape()[0], dtype, device);
        device.backend().CallKernel<SolveKernel>(a.Transpose(), b, out);
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(InverseKernel, CudaInverseKernel);

}  // namespace cuda
}  // namespace chainerx
