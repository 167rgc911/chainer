#include "chainerx/cuda/cuda_device.h"

#include <cstdint>
#include <mutex>
#include <type_traits>

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cuda_fp16.hpp>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/backend.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/hipblas.h"
#include "chainerx/cuda/cusolver.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/float16.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/linalg.h"
#include "chainerx/kernels/math.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/macro.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/math.h"

namespace chainerx {
namespace cuda {

class CudaQRKernel : public QRKernel {
public:
    std::tuple<Array, Array> Call(const Array& a) override {
        Device& device = a.device();
        Dtype dtype = internal::GetMathResultDtype(a.dtype());
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);

        throw NotImplementedError("QR decomposition is not yet implemented for cuda device");

    }
};

CHAINERX_CUDA_REGISTER_KERNEL(QRKernel, CudaQRKernel);

}  // namespace cuda
}  // namespace chainerx
