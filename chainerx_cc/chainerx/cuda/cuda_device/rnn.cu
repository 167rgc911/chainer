#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <algorithm>
#include <cstdint>
#include <functional>
#include <memory>
#include <tuple>
#include <utility>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <absl/types/optional.h>

#include "chainerx/array.h"
#include "chainerx/backend_util.h"
#include "chainerx/constant.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_backend.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/hipDNN.h"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/kernels/connection.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/kernels/rnn.h"
#include "chainerx/macro.h"
#include "chainerx/native/kernel_regist.h"
#include "chainerx/routines/connection.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/manipulation.h"
#include "chainerx/shape.h"
#include "chainerx/stack_vector.h"

namespace chainerx {
namespace cuda {
namespace {

__global__ void initGPUData_ker(float* data, int numElements, float* value) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numElements) {
        data[tid] = value[tid];
    }
}

void initGPUData(float* data, int numElements, float* value) {
    dim3 gridDim;
    dim3 blockDim;
    blockDim.x = 1024;
    gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;
    initGPUData_ker<<<gridDim, blockDim>>>(data, numElements, value);
}

void weights_forward(
        cuda_internal::DeviceInternals& device_internals,
        hipdnnRNNDescriptor_t rnn_desc,
        const std::vector<std::vector<Array>> ws,
        const std::vector<std::vector<Array>> bs,
        int n_layers,
        int num_directions,
        hipdnnTensorDescriptor_t x_desc,
        hipdnnFilterDescriptor_t w_desc,
        Array& w) {
    for (int layer = 0; layer < n_layers; layer++) {
        for (int8_t di = 0; di < num_directions; di++) {
            for (uint lin_layer_id = 0; lin_layer_id < ws[0].size(); lin_layer_id++) {
                int64_t index = num_directions * layer + di;
                hipdnnFilterDescriptor_t linlayermatdesc;
                hipdnnCreateFilterDescriptor(&linlayermatdesc);
                float* m_offset;
                device_internals.cudnn_handle().Call(
                        hipdnnGetRNNLinLayerMatrixParams,
                        rnn_desc,
                        index,
                        x_desc,
                        w_desc,
                        internal::GetRawOffsetData(w),
                        lin_layer_id,
                        linlayermatdesc,
                        reinterpret_cast<void**>(&m_offset));
                hipdnnDataType_t dataType;
                hipdnnTensorFormat_t format;
                int nbDims;
                int filterDimA[3];

                hipdnnGetFilterNdDescriptor(linlayermatdesc, 3, &dataType, &format, &nbDims, filterDimA);
                Array w_temp = AsContiguous(ws[index][lin_layer_id].AsType(Dtype::kFloat32));
                initGPUData(
                        m_offset,
                        filterDimA[0] * filterDimA[1] * filterDimA[2],
                        reinterpret_cast<float*>(internal::GetRawOffsetData(w_temp)));
                hipdnnDestroyFilterDescriptor(linlayermatdesc);

                hipdnnFilterDescriptor_t linlayerbiasdesc;
                hipdnnCreateFilterDescriptor(&linlayerbiasdesc);
                float* b_offset;
                device_internals.cudnn_handle().Call(
                        hipdnnGetRNNLinLayerBiasParams,
                        rnn_desc,
                        index,
                        x_desc,
                        w_desc,
                        internal::GetRawOffsetData(w),
                        lin_layer_id,
                        linlayerbiasdesc,
                        reinterpret_cast<void**>(&b_offset));
                hipdnnGetFilterNdDescriptor(linlayerbiasdesc, 3, &dataType, &format, &nbDims, filterDimA);
                Array b_temp = AsContiguous(bs[index][lin_layer_id].AsType(Dtype::kFloat32));
                initGPUData(
                        b_offset,
                        filterDimA[0] * filterDimA[1] * filterDimA[2],
                        reinterpret_cast<float*>(internal::GetRawOffsetData(b_temp)));
                hipdnnDestroyFilterDescriptor(linlayerbiasdesc);
            }
        }
    }
}

std::vector<Array> weights_backward(
        CudaDevice& device,
        hipdnnRNNDescriptor_t& rnn_desc,
        hipdnnTensorDescriptor_t dummy_x_desc,
        hipdnnFilterDescriptor_t w_desc,
        Array w,
        std::vector<std::vector<Array>> ws,
        std::vector<std::vector<Array>> bs,
        int64_t n_layers,
        int64_t num_directions,
        Dtype type) {
    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(device);
    std::vector<Array> ret;
    for (int64_t layer = 0; layer < n_layers; layer++) {
        for (int8_t di = 0; di < num_directions; di++) {
            for (uint lin_layer_id = 0; lin_layer_id < ws[0].size(); lin_layer_id++) {
                int64_t index = num_directions * layer + di;
                hipdnnFilterDescriptor_t linlayermatdesc;
                hipdnnCreateFilterDescriptor(&linlayermatdesc);
                float* m_offset;
                device_internals.cudnn_handle().Call(
                        hipdnnGetRNNLinLayerMatrixParams,
                        rnn_desc,
                        index,
                        dummy_x_desc,
                        w_desc,
                        internal::GetRawOffsetData(w),
                        lin_layer_id,
                        linlayermatdesc,
                        reinterpret_cast<void**>(&m_offset));
                Array m = AsContiguous(Zeros(ws[index][lin_layer_id].shape(), type, ws[index][lin_layer_id].device()));
                hipdnnDataType_t dataType;
                hipdnnTensorFormat_t format;
                int nbDims;
                int filterDimA[3];

                hipdnnGetFilterNdDescriptor(linlayermatdesc, 3, &dataType, &format, &nbDims, filterDimA);
                initGPUData(
                        reinterpret_cast<float*>(internal::GetRawOffsetData(m)), filterDimA[0] * filterDimA[1] * filterDimA[2], m_offset);
                hipdnnDestroyFilterDescriptor(linlayermatdesc);
                ret.push_back(m);
                hipdnnFilterDescriptor_t linlayerbiasdesc;
                hipdnnCreateFilterDescriptor(&linlayerbiasdesc);
                float* b_offset;
                device_internals.cudnn_handle().Call(
                        hipdnnGetRNNLinLayerBiasParams,
                        rnn_desc,
                        index,
                        dummy_x_desc,
                        w_desc,
                        internal::GetRawOffsetData(w),
                        lin_layer_id,
                        linlayerbiasdesc,
                        reinterpret_cast<void**>(&b_offset));
                Array b = AsContiguous(Zeros(bs[index][lin_layer_id].shape(), type, bs[index][lin_layer_id].device()));
                hipdnnGetFilterNdDescriptor(linlayerbiasdesc, 3, &dataType, &format, &nbDims, filterDimA);
                initGPUData(
                        reinterpret_cast<float*>(internal::GetRawOffsetData(b)), filterDimA[0] * filterDimA[1] * filterDimA[2], b_offset);
                hipdnnDestroyFilterDescriptor(linlayerbiasdesc);
                ret.push_back(b);
            }
        }
    }
    return ret;
}

class CudaRnnKernel : public RnnKernel {
public:
    std::tuple<std::vector<std::vector<Array>>, std::unique_ptr<chainerx::RnnGradState>> Call(
            int64_t n_layers,
            Array hx,
            absl::optional<Array> cx,
            const std::vector<std::vector<Array>>& ws,
            const std::vector<std::vector<Array>>& bs,
            const std::vector<Array>& xs,
            const int8_t bidirectional,
            const int8_t mode) override {
        CudaDevice& device = dynamic_cast<CudaDevice&>(hx.device());
        CudaSetDeviceScope scope{device.index()};
        auto& backend = static_cast<CudaBackend&>(device.backend());  // NOLINT
        Dtype type = hx.dtype();
        cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(device);

        const auto input_dim = xs[0].shape()[1];
        const auto hidden_dim = hx.shape()[2];
        const auto num_directions = bidirectional == 1 ? 2 : 1;
        const auto num_layers = n_layers;
        const auto rnn_direction = bidirectional == 1 ? HIPDNN_BIDIRECTIONAL : HIPDNN_UNIDIRECTIONAL;
        const auto rnn_mode = mode == 1 ? HIPDNN_LSTM : HIPDNN_GRU;
        const auto rnn_input = HIPDNN_LINEAR_INPUT;
        hipdnnRNNDescriptor_t rnn_desc;
        uint64_t seed = 1337ull;

        hipdnnDropoutDescriptor_t dropoutDesc;
        hipdnnCreateDropoutDescriptor(&dropoutDesc);

        size_t stateSize;
        void* states;
        device_internals.cudnn_handle().Call(hipdnnDropoutGetStatesSize, &stateSize);

        hipMalloc(&states, stateSize);

        hipdnnSetDropoutDescriptor(dropoutDesc, device_internals.cudnn_handle().handle(), 0, states, stateSize, seed);
        hipdnnCreateRNNDescriptor(&rnn_desc);
        device_internals.cudnn_handle().Call(
                hipdnnSetRNNDescriptor,
                rnn_desc,
                hidden_dim,
                num_layers,
                dropoutDesc,
                rnn_input,
                rnn_direction,
                rnn_mode,
                HIPDNN_RNN_ALGO_STANDARD,
                HIPDNN_DATA_FLOAT);
        hipdnnTensorDescriptor_t *x_desc, *y_desc;
        x_desc = reinterpret_cast<hipdnnTensorDescriptor_t*>(malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t)));
        y_desc = reinterpret_cast<hipdnnTensorDescriptor_t*>(malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t)));
        std::vector<cuda_internal::CudnnTensorDescriptor> xs_desc;
        std::vector<cuda_internal::CudnnTensorDescriptor> ys_desc;
        std::vector<Array> ys;
        std::vector<Array> xs_cont;
        for (uint i = 0; i < xs.size(); i++) {
            Shape xs_shape{xs[i].shape()[0], xs[i].shape()[1], 1};
            Shape ys_shape{xs[i].shape()[0], num_directions * hidden_dim, 1};
            xs_cont.push_back(AsContiguous(xs[i].AsType(Dtype::kFloat32)));
            ys.push_back(
                    AsContiguous(Zeros({xs_cont[i].shape()[0], num_directions * hidden_dim}, xs_cont[i].dtype(), xs_cont[i].device())));
            xs_desc.push_back(cuda_internal::CudnnTensorDescriptor(Reshape(xs_cont[i], xs_shape)));
            ys_desc.push_back(cuda_internal::CudnnTensorDescriptor(Reshape(ys[i], ys_shape)));
            x_desc[i] = *xs_desc[i];
            y_desc[i] = *ys_desc[i];
        }
        Array x = Concatenate(xs_cont, 0);
        Array y = Concatenate(ys, 0);

        size_t weight_size;
        device_internals.cudnn_handle().Call(hipdnnGetRNNParamsSize, rnn_desc, x_desc[0], &weight_size, HIPDNN_DATA_FLOAT);
        Array w = AsContiguous(Zeros({static_cast<int>(weight_size) / 4, 1, 1}, x.dtype(), x.device()));
        cuda_internal::CudnnFilterDescriptor wDesc{w};

        weights_forward(device_internals, rnn_desc, ws, bs, n_layers, num_directions, x_desc[0], *wDesc, w);
        size_t workSize;
        size_t reserve_size;
        device_internals.cudnn_handle().Call(hipdnnGetRNNWorkspaceSize, rnn_desc, xs.size(), x_desc, &workSize);
        Array workspace = AsContiguous(Zeros({static_cast<int64_t>(workSize)}, hx.dtype(), hx.device()));

        device_internals.cudnn_handle().Call(hipdnnGetRNNTrainingReserveSize, rnn_desc, xs.size(), x_desc, &reserve_size);
        Array reserve = AsContiguous(Zeros({static_cast<int64_t>(reserve_size)}, hx.dtype(), hx.device()));
        hx = AsContiguous(hx.AsType(Dtype::kFloat32));
        Array hy = AsContiguous(Zeros(hx.shape(), hx.dtype(), hx.device()));
        Array cy = AsContiguous(Zeros(hx.shape(), hx.dtype(), hx.device()));
        cuda_internal::CudnnTensorDescriptor hxDesc{hx};
        Array _cx;
        if (cx.has_value()) {
            _cx = AsContiguous((*cx).AsType(Dtype::kFloat32));
        } else {
            _cx = Zeros(hx.shape(), hx.dtype(), hx.device());
        }
        cuda_internal::CudnnTensorDescriptor cxDesc{_cx};
        cuda_internal::CudnnTensorDescriptor hyDesc{hy};
        cuda_internal::CudnnTensorDescriptor cyDesc{cy};
        device_internals.cudnn_handle().Call(
                hipdnnRNNForwardTraining,
                rnn_desc,
                xs.size(),
                x_desc,
                internal::GetRawOffsetData(x),
                *hxDesc,
                internal::GetRawOffsetData(hx),
                *cxDesc,
                internal::GetRawOffsetData(_cx),
                *wDesc,
                internal::GetRawOffsetData(w),
                y_desc,
                internal::GetRawOffsetData(y),
                *hyDesc,
                internal::GetRawOffsetData(hy),
                *cyDesc,
                internal::GetRawOffsetData(cy),
                internal::GetRawOffsetData(workspace),
                workSize,
                internal::GetRawOffsetData(reserve),
                reserve_size);
        std::vector<int64_t> split_indices;
        for (uint i = 0; i < xs.size() - 1; i++) {
            if (i != 0) {
                split_indices.push_back(split_indices[i - 1] + xs[i].shape()[0]);
            } else {
                split_indices.push_back(xs[i].shape()[0]);
            }
        }

        std::unique_ptr<RnnGradState> state = std::make_unique<GenericRnnGradState>(rnn_desc, *wDesc, w, reserve, workspace);
        y = y.AsType(type);
        ys = Split(y, split_indices, 0);

        std::vector<Array> out_states;
        out_states.push_back(hy.AsType(type));
        if (cx.has_value()) {
            out_states.push_back(cy.AsType(type));
        }
        std::vector<std::vector<Array>> ret;
        ret.push_back(out_states);
        ret.push_back(ys);
        return std::make_tuple(std::move(ret), std::move(state));
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(RnnKernel, CudaRnnKernel);

class CudaRnnBackwardKernel : public RnnBackwardKernel {
public:
    std::vector<std::vector<Array>> Call(
            int64_t n_layers,
            Array hx,
            absl::optional<Array> cx,
            const std::vector<std::vector<Array>>& ws,
            const std::vector<std::vector<Array>>& bs,
            const std::vector<Array>& xs,
            Array dhy,
            absl::optional<Array> dcy,
            std::vector<Array> ys,
            std::vector<Array> dys,
            const int8_t bidirectional,
            const std::shared_ptr<chainerx::RnnGradState>& state) override {
        CudaDevice& device = dynamic_cast<CudaDevice&>(hx.device());
        CudaSetDeviceScope scope{device.index()};

        cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(device);
        auto cuda_state = dynamic_cast<GenericRnnGradState&>(*state);
        Dtype type = hx.dtype();
        const auto input_dim = xs[0].shape()[1];
        const auto hidden_dim = hx.shape()[2];
        const auto num_directions = bidirectional == 1 ? 2 : 1;

        hipdnnRNNDescriptor_t rnn_desc = cuda_state.rnn_desc();
        std::vector<Array> dxs;
        std::vector<cuda_internal::CudnnTensorDescriptor> xsDesc, dxsDesc, ysDesc, dysDesc;
        hipdnnTensorDescriptor_t* xs_desc = reinterpret_cast<hipdnnTensorDescriptor_t*>(malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t)));
        hipdnnTensorDescriptor_t* dxs_desc = reinterpret_cast<hipdnnTensorDescriptor_t*>(malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t)));
        hipdnnTensorDescriptor_t* ys_desc = reinterpret_cast<hipdnnTensorDescriptor_t*>(malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t)));
        hipdnnTensorDescriptor_t* dys_desc = reinterpret_cast<hipdnnTensorDescriptor_t*>(malloc(xs.size() * sizeof(hipdnnTensorDescriptor_t)));
        std::vector<Array> xs_cont;
        for (uint i = 0; i < xs.size(); i++) {
            Shape xs_shape{xs[i].shape()[0], xs[i].shape()[1], 1};
            Shape ys_shape{ys[i].shape()[0], ys[i].shape()[1], 1};

            xs_cont.push_back(AsContiguous(xs[i].AsType(Dtype::kFloat32)));
            ys[i] = AsContiguous(ys[i].AsType(Dtype::kFloat32));
            dys[i] = AsContiguous(dys[i].AsType(Dtype::kFloat32));
            xsDesc.push_back(cuda_internal::CudnnTensorDescriptor(Reshape(xs_cont[i], xs_shape)));
            xs_desc[i] = *xsDesc[i];
            dxs.push_back(AsContiguous(Zeros(xs_cont[i].shape(), xs_cont[i].dtype(), xs_cont[i].device())));
            dxsDesc.push_back(cuda_internal::CudnnTensorDescriptor(Reshape(dxs[i], xs_shape)));
            dxs_desc[i] = *dxsDesc[i];
            ysDesc.push_back(cuda_internal::CudnnTensorDescriptor(Reshape(ys[i], ys_shape)));
            ys_desc[i] = *ysDesc[i];
            dysDesc.push_back(cuda_internal::CudnnTensorDescriptor(Reshape(dys[i], ys_shape)));
            dys_desc[i] = *dysDesc[i];
        }
        Array dx = AsContiguous(Concatenate(dxs, 0));
        Array x = AsContiguous(Concatenate(xs_cont, 0));
        Array y = AsContiguous(Concatenate(ys, 0));
        Array dy = AsContiguous(Concatenate(dys, 0));
        hipdnnFilterDescriptor_t wDesc = cuda_state.wDesc();
        Array w = AsContiguous(cuda_state.w());
        Array reserve = AsContiguous(cuda_state.reserve());
        Array workspace = AsContiguous(cuda_state.workspace());
        size_t reserve_size = reserve.shape()[0];
        size_t workSize = workspace.shape()[0];
        hx = AsContiguous(hx.AsType(Dtype::kFloat32));
        dhy = AsContiguous(dhy.AsType(Dtype::kFloat32));

        cuda_internal::CudnnTensorDescriptor hx_desc{hx};
        Array _cx;
        Array _dcy;
        if (cx.has_value()) {
            _cx = AsContiguous((*cx).AsType(Dtype::kFloat32));
            _dcy = AsContiguous((*dcy).AsType(Dtype::kFloat32));
        } else {
            _cx = Zeros(hx.shape(), hx.dtype(), hx.device());
            _dcy = Zeros(hx.shape(), hx.dtype(), hx.device());
        }
        cuda_internal::CudnnTensorDescriptor cx_desc{_cx};
        cuda_internal::CudnnTensorDescriptor dcy_desc{_dcy};
        cuda_internal::CudnnTensorDescriptor dhy_desc{dhy};
        Array dhx = AsContiguous(Zeros(hx.shape(), hx.dtype(), hx.device()));
        cuda_internal::CudnnTensorDescriptor dhx_desc{dhx};
        Array dcx = AsContiguous(Zeros(hx.shape(), hx.dtype(), hx.device()));
        cuda_internal::CudnnTensorDescriptor dcx_desc{dcx};
        reserve = AsContiguous(reserve);

        device_internals.cudnn_handle().Call(
                hipdnnRNNBackwardData,
                rnn_desc,
                xs.size(),
                ys_desc,
                internal::GetRawOffsetData(y),
                dys_desc,
                internal::GetRawOffsetData(dy),
                *dhy_desc,
                internal::GetRawOffsetData(dhy),
                *dcy_desc,
                internal::GetRawOffsetData(_dcy),
                wDesc,
                internal::GetRawOffsetData(w),
                *hx_desc,
                internal::GetRawOffsetData(hx),
                *cx_desc,
                internal::GetRawOffsetData(_cx),
                dxs_desc,
                internal::GetRawOffsetData(dx),
                *dhx_desc,
                internal::GetRawOffsetData(dhx),
                *dcx_desc,
                internal::GetRawOffsetData(dcx),
                internal::GetRawOffsetData(workspace),
                workSize,
                internal::GetRawOffsetData(reserve),
                reserve_size);

        Array dw = AsContiguous(Zeros(w.shape(), hx.dtype(), hx.device()));
        cuda_internal::CudnnFilterDescriptor dwDesc{dw};
        device_internals.cudnn_handle().Call(
                hipdnnRNNBackwardWeights,
                rnn_desc,
                xs.size(),
                xs_desc,
                internal::GetRawOffsetData(x),
                *hx_desc,
                internal::GetRawOffsetData(hx),
                ys_desc,
                internal::GetRawOffsetData(y),
                internal::GetRawOffsetData(workspace),
                workSize,
                *dwDesc,
                internal::GetRawOffsetData(dw),
                internal::GetRawOffsetData(reserve),
                reserve_size);

        std::vector<int64_t> split_indices;
        for (uint i = 0; i < xs.size() - 1; i++) {
            if (i != 0) {
                split_indices.push_back(split_indices[i - 1] + xs[i].shape()[0]);
            } else {
                split_indices.push_back(xs[i].shape()[0]);
            }
        }
        dx = dx.AsType(type);
        dxs = Split(dx, split_indices, 0);
        std::vector<Array> dstate;
        dstate.push_back(dhx.AsType(type));
        if (cx.has_value()) {
            dstate.push_back(dcx.AsType(type));
        }
        std::vector<std::vector<Array>> ret;
        ret.push_back(dstate);
        ret.push_back(weights_backward(device, rnn_desc, dxs_desc[0], *dwDesc, dw, ws, bs, n_layers, num_directions, type));
        ret.push_back(dxs);
        return ret;
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(RnnBackwardKernel, CudaRnnBackwardKernel);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
